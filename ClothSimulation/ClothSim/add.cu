#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "add.cuh"

__global__
void add(int n, float* x, float* y) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i += stride) {
		y[i] = x[i] + y[i];
	}
}

Add::Add() 
{

}

Add::~Add()
{

}

void Add::AddVertByRand()
{
	size_t tmpVertexPointerSize;
	float3* tmpVertexPointer;
	hipGraphicsMapResources(1, &vertexBuf, 0);
	hipGraphicsResourceGetMappedPointer((void **)&tmpVertexPointer, &tmpVertexPointerSize, vertexBuf);


}
