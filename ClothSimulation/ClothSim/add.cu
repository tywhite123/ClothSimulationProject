#include "hip/hip_runtime.h"

#include ""

#include "add.cuh"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

__global__
void add(unsigned int size, float time, float3* vertexBuf) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= size)
		return;
	
	float3 pos = vertexBuf[index];

	hiprandState_t state;

	hiprand_init(0, 0, 0, &state);

	int randomVal = (hiprand(&state) % 3);

	pos.y = sin(time * 0.001f + blockIdx.x * threadIdx.x * 0.5f) * 10;

	vertexBuf[index] = pos;



}


__global__
void Integrate(unsigned int size, float time, float3* vertexBuf, float3 grav, float damping, float3* oldPositions) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= size)
		return;

	float3 pos = vertexBuf[index];
	float3 old = oldPositions[index];
	float3 temp = pos;
	 
	//pos.x = pos.x + damping * (pos.x-old.x) + grav.x * (time*time);
	if (pos.y >= -5000.0f) {
		pos.y = pos.y + damping * (pos.y-old.y) + grav.y * (time*time);
	}
	
	//pos.z = pos.z + damping * (pos.z-old.z) + grav.z * (time*time);
	//pos = pos  + (pos - old) * damping + grav * (time*time);

	/*if (pos.y < 0.0f) {
		pos.y = 0.0f;
	}*/


	vertexBuf[index] = pos;
	oldPositions[index] = temp;



}

__global__
void FloorConstraint(float3* vertexBuf, float floor, float size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= size)
		return;

	float3 pos = vertexBuf[index];

	if (pos.y < floor) {
		pos.y = floor;
	}

	vertexBuf[index] = pos;

}


__global__
void SphereConstraint(float3* vertexBuf, float3 spherePoint, float size, float radius) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= size)
		return;

	float3 pos = vertexBuf[index];

	float3 delta;
	delta.x = pos.x - spherePoint.x;
	delta.y = pos.y - spherePoint.y;
	delta.z = pos.z - spherePoint.z;

	float dist = sqrt((delta.x*delta.x)+(delta.y*delta.y)+(delta.z*delta.z));

	if (dist < radius) {
		pos.x = spherePoint.x + delta.x *(radius / dist);
		pos.y = spherePoint.y + delta.y *(radius / dist);
		pos.z = spherePoint.z + delta.z *(radius / dist);

	}


	vertexBuf[index] = pos;

}

Add::Add(unsigned int size)
{
	hipMalloc((void**)&oldPositions, size * sizeof(float3));
}

Add::~Add()
{
	hipFree(oldPositions);
}

void Add::BindBuffers(HeightMap * map)
{
	hipGraphicsGLRegisterBuffer(&vertexBuf, map->getVertexBuffer(), cudaGraphicsMapFlagsNone);

	//dim3 block(256, 1, 1);
	////dim3 grid((size + block.x - 1) / block.x, 1, 1);
}

void Add::AddByRand(unsigned int size, float time)
{
	std::size_t tmpVertexPointerSize;
	float3* tmpVertexPointer;
	hipGraphicsMapResources(1, &vertexBuf, 0);
	hipGraphicsResourceGetMappedPointer((void**)&tmpVertexPointer, &tmpVertexPointerSize, vertexBuf);

	dim3 block(256, 1, 1);
	dim3 grid((size + block.x - 1) / block.x, 1, 1);

	add << <grid, block >> > (size, time, tmpVertexPointer);
	
	hipGraphicsUnmapResources(1, &vertexBuf, 0);

}

void Add::IntergrateTest(unsigned int size, float time, float damping, Vector3 gravity)
{
	std::size_t tmpVertexPointerSize;
	float3* tmpVertexPointer;
	hipGraphicsMapResources(1, &vertexBuf, 0);
	hipGraphicsResourceGetMappedPointer((void**)&tmpVertexPointer, &tmpVertexPointerSize, vertexBuf);

	dim3 block(256, 1, 1);
	dim3 grid((size + block.x - 1) / block.x, 1, 1);

	float3 grav;
	grav.x = gravity.x;
	grav.y = gravity.y;
	grav.z = gravity.z;

	//-2000.0f, 2000.0f, -2000.0f
	float3 spherePoint;
	spherePoint.x = 2000.0f;
	spherePoint.y = -2000.0f;
	spherePoint.z = 2000.0f;

	Integrate << <grid, block >> > (size, 0.25, tmpVertexPointer, grav, damping, oldPositions);
	//FloorConstraint << <grid, block >> > (tmpVertexPointer, 0.0f, size);
	SphereConstraint << <grid, block >> > (tmpVertexPointer, spherePoint, size, 1010);

	hipGraphicsUnmapResources(1, &vertexBuf, 0);

}
