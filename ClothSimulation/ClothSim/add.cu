#include "hip/hip_runtime.h"

#include ""

#include "add.cuh"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

__global__
void add(unsigned int size, float time, float3* vertexBuf) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= size)
		return;
	
	float3 pos = vertexBuf[index];

	hiprandState_t state;

	hiprand_init(0, 0, 0, &state);

	int randomVal = (hiprand(&state) % 3);

	pos.y = sin(time * 0.001f + blockIdx.x * threadIdx.x * 0.5f) * 10;

	vertexBuf[index] = pos;



}


__global__
void Integrate(unsigned int size, float time, float3* vertexBuf, float3 grav, float damping) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= size)
		return;

	float3 pos = vertexBuf[index];
	float3 oldPos = pos;

	//pos.x = pos.x + damping + grav.x*(time*time);
	pos.y = pos.y - damping + grav.y*(time*time);
	//pos.y = pos.y + damping + grav.z*(time*time);

	vertexBuf[index] = pos;



}


Add::Add() 
{
}

Add::~Add()
{

}

void Add::BindBuffers(HeightMap * map)
{
	hipGraphicsGLRegisterBuffer(&vertexBuf, map->getVertexBuffer(), cudaGraphicsMapFlagsNone);

	//dim3 block(256, 1, 1);
	////dim3 grid((size + block.x - 1) / block.x, 1, 1);
}

void Add::AddByRand(unsigned int size, float time)
{
	std::size_t tmpVertexPointerSize;
	float3* tmpVertexPointer;
	hipGraphicsMapResources(1, &vertexBuf, 0);
	hipGraphicsResourceGetMappedPointer((void**)&tmpVertexPointer, &tmpVertexPointerSize, vertexBuf);

	dim3 block(256, 1, 1);
	dim3 grid((size + block.x - 1) / block.x, 1, 1);

	add << <grid, block >> > (size, time, tmpVertexPointer);
	
	hipGraphicsUnmapResources(1, &vertexBuf, 0);

}

void Add::IntergrateTest(unsigned int size, float time, float damping, Vector3 gravity)
{
	std::size_t tmpVertexPointerSize;
	float3* tmpVertexPointer;
	hipGraphicsMapResources(1, &vertexBuf, 0);
	hipGraphicsResourceGetMappedPointer((void**)&tmpVertexPointer, &tmpVertexPointerSize, vertexBuf);

	dim3 block(256, 1, 1);
	dim3 grid((size + block.x - 1) / block.x, 1, 1);

	float3 grav;
	grav.x = gravity.x;
	grav.y = gravity.y;
	grav.z = gravity.z;

	Integrate << <grid, block >> > (size, 0.01, tmpVertexPointer, grav, damping);

	hipGraphicsUnmapResources(1, &vertexBuf, 0);

}
